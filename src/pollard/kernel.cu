#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"
#include "primegen.h"

#define NUM_BLOCKS 1
#define THREADS_PER_BLOCK 1

#define B_START 2
#define TABLE_SIZE (200 * 1000 * 1000)

__global__
void parallel_factorize_kernel(UL N, unsigned *primes, bool *finished,
                               mpz_t *result) {
  mpz_t n, a, d, p, e, b, tmp, tmp_2, MPZ_ONE;

  mpz_init(&n);
  mpz_init(&a);
  mpz_init(&d);
  mpz_init(&p);
  mpz_init(&e);
  mpz_init(&b);
  mpz_init(&tmp);
  mpz_init(&tmp_2);

  mpz_init(&MPZ_ONE);
  mpz_set_i(&MPZ_ONE, 1);

  mpz_set_lui(&n, N);

  int count = 0;

  unsigned B;
  unsigned max_B = ((N < TABLE_SIZE) ? N : TABLE_SIZE);
  for (B = B_START; B < max_B; B *= 2) {
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned threads = gridDim.x * blockDim.x;
    unsigned i = blockIdx.x * blockDim.x;

    unsigned it;
    unsigned max_it = 80;

    mpz_set_lui(&e, (UL) 1);
    unsigned p_i;
    for (p_i = tid; primes[p_i] < B; p_i += threads) {
      mpz_set_lui(&p, (UL) primes[p_i]);
      // TODO: replace MPZ_ONE with logB / logp
      mpz_set_lui(&tmp_2, (UL) (log((double) B) / log((double) primes[p_i])));
      // tmp_2 = floor(log b / log p)
      mpz_powmod(&tmp, &p, &tmp_2, &n); // tmp = (p ** tmp_2) % n
      mpz_mult(&tmp_2, &tmp, &e);       // tmp_2 = tmp * e
      mpz_set(&e, &tmp_2);              // e = tmp_2
    }

    if (mpz_equal(&e, &MPZ_ONE)) return;

    // char *e_str = mpz_get_str(&e, NULL, 0);
    // printf("\tUsing e = %s\n", e_str);
    // free(e_str);

    // try a variety of a values
    mpz_set_lui(&a, 2 + tid);
    for (it = 0; it < max_it; it ++) {
      count ++;
      if (*finished) {
        printf("Ran in %d iterations.\n", count);
        return;
      }
      // char *a_str = mpz_get_str(&a, NULL, 0);
      // printf("\t\tUsing a = %s\n", a_str);
      // free(a_str);
      // check for a freebie
      mpz_gcd(&d, &a, &n);
      if (mpz_lt(&MPZ_ONE, &d)) {
        *result = d;
        *finished = true;
        printf("Ran in %d iterations.\n", count);
        return;
      }

      mpz_powmod(&b, &a, &e, &n);  // b = (a ** e) % n
      mpz_sub(&tmp, &b, &MPZ_ONE); // tmp = b - 1
      mpz_gcd(&d, &tmp, &n);       // d = gcd(tmp, n)

      // success!
      if (mpz_lt(&MPZ_ONE, &d) && mpz_lt(&d, &n)) {
        *result = d;
        *finished = true;
        printf("Ran in %d iterations.\n", count);
        return;
      }

      // otherwise get a new value for a
      // mpz_mult(&tmp, &a, &a);               // tmp = a ** 2
      // mpz_set_lui(&a, (UL) (i + it + tid)); // a = i + it + tid
      // mpz_add(&tmp_2, &tmp, &a);            // tmp_2 = &tmp + a
      // mpz_div(&tmp, &a, &tmp_2, &n);        // a = tmp_2 % n
      mpz_add(&tmp, &a, &MPZ_ONE);
      mpz_set(&a, &tmp);
    }
  }
  // couldn't find anything... :(
  printf("Ran in %d iterations (and failed).\n", count);
}

int factorize(UL n, unsigned *primes, mpz_t *factor) {
  unsigned blocks = NUM_BLOCKS;
  unsigned threads_per_block = THREADS_PER_BLOCK;
  unsigned threads = blocks * threads_per_block;

  size_t result_bytes = sizeof(mpz_t);

  mpz_t *d_result;
  bool *d_finished;
  if ((hipSuccess != hipMalloc((void **) &d_result, result_bytes)) ||
      (hipSuccess != hipMalloc((void **) &d_finished, sizeof(bool))) ||
      (hipSuccess != hipMemset(d_result, 0L, result_bytes)) ||
      (hipSuccess != hipMemset(d_finished, false, sizeof(bool)))) {
    fprintf(stderr, "Unable to allocate device memory!\n");
    return -1;
  }

  parallel_factorize_kernel<<<blocks, threads_per_block>>>
    (n, primes, d_finished, d_result);

  if (hipSuccess != hipMemcpy(factor, d_result, result_bytes,
                                hipMemcpyDeviceToHost)) {
    fprintf(stderr, "Unable to retrieve result from host!\n");
    return -1;
  }

  hipFree(primes);
  hipFree(d_result);
  hipFree(d_finished);

  return -1;
}

void get_prime_table(unsigned *table, unsigned n) {
  primegen pg;

  primegen_init(&pg);

  for (unsigned i = 0; i < n; i ++) {
    table[i] = (unsigned) primegen_next(&pg);
  }
}

int generate_prime_table(unsigned **d_table) {
  /* approximately the number of unsigned 32-bit primes       */
  /* (actual number is 203,280,221)                           */
  /* paper claimed to use ~170,000,000 primes for experiments */
  /* may want to write this to disk at some point...          */
  unsigned primes = TABLE_SIZE;
  unsigned *h_table = (unsigned *) malloc(primes * sizeof(unsigned));
  if (NULL == h_table) {
    fprintf(stderr, "Unable to allocate host prime table!\n");
    return -1;
  }

  printf("Generating prime table... "); fflush(stdout);
  get_prime_table(h_table, primes);
  printf("done!\n"); fflush(stdout);

  /* move prime table to the gpu */
  printf("Transferring table to the gpu... "); fflush(stdout);
  if ((hipSuccess != hipMalloc((void **) d_table,
                                 primes * sizeof(unsigned))) ||
      (hipSuccess != hipMemcpy((void *) *d_table, (void *) h_table,
                                 primes * sizeof(unsigned),
                                 hipMemcpyHostToDevice))) {
    fprintf(stderr, "Unable to allocate device prime table!\n");
    return -1;
  }
  printf("done!\n"); fflush(stdout);

  free(h_table);

  return 0;
}
