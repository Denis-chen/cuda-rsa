#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"
#include "primegen.h"

#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 32

#define B_START 2
#define TABLE_SIZE (200 * 1000 * 1000)

__global__ 
void trial_division_kernel(mpz_t n, unsigned *primes, bool *finished,
                               mpz_t *result) {
  unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned threads = gridDim.x * blockDim.x;
  mpz_t div;
  mpz_t mod;
  mpz_t factor;
  mpz_t zero;
  mpz_t cap;
  mpz_t num_threads;

  mpz_init(&div);
  mpz_init(&mod);
  mpz_init(&factor);
  mpz_init(&zero);
  mpz_init(&cap);
  mpz_init(&num_threads);

  mpz_set_lui(&zero, 0);
  mpz_set_lui(&num_threads, threads);
  mpz_set_lui(&factor, tid + 2);

  mpz_set(&cap, &n); // should be sqrt n

  while (mpz_lte(&factor, &cap)) {
    mpz_div(&div, &mod, &n, &factor);

    if (mpz_equal(&mod, &zero)) {
      *result = factor;
      *finished = true;
      return;
    }

  if (*finished) return;

    mpz_add(&div, &factor, &num_threads);
    mpz_set(&factor, &div);
  }
}

__global__ 
void prime_division_kernel(mpz_t n, unsigned *primes, bool *finished,
                               mpz_t *result) {
  unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned threads = gridDim.x * blockDim.x;
  unsigned i;
  mpz_t div;
  mpz_t mod;
  mpz_t factor;
  mpz_t zero;
  mpz_t cap;
  mpz_t num_threads;

  mpz_init(&div);
  mpz_init(&mod);
  mpz_init(&factor);
  mpz_init(&zero);
  mpz_init(&cap);
  mpz_init(&num_threads);

  mpz_set_lui(&zero, 0);
  mpz_set_lui(&num_threads, threads);
  mpz_set_lui(&factor, tid + 2);

  mpz_set(&cap, &n); // should be sqrt n

  i = tid;
  while (true) {
    unsigned long p = primes[i];
    mpz_set_lui(&factor, p);
    if (mpz_gt(&factor, &n)) break;

    printf("Trying division by %d\n", p);

    mpz_div(&div, &mod, &n, &factor);

    if (mpz_equal(&mod, &zero)) {
      printf("got it\n");
      *result = factor;
      *finished = true;
      return;
    }

    if (*finished) break;

    i += threads;
  }
}
__global__
void parallel_factorize_kernel(mpz_t n, unsigned *primes, bool *finished,
                               mpz_t *result) {
  unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned threads = gridDim.x * blockDim.x;
  unsigned i = blockIdx.x * blockDim.x;

  mpz_t a, d, p, e, b, tmp, tmp_2, MPZ_ONE;
  mpz_init(&a);
  mpz_init(&d);
  mpz_init(&p);
  mpz_init(&e);
  mpz_init(&b);
  mpz_init(&tmp);
  mpz_init(&tmp_2);

  mpz_init(&MPZ_ONE);
  mpz_set_i(&MPZ_ONE, 1);

  int count = 0;

  unsigned B;
  const unsigned B_MAX = TABLE_SIZE;

  for (B = B_START; B < B_MAX; B *= 2) {
    unsigned it;
    unsigned max_it = 80;
    unsigned p_i;

    mpz_set_lui(&e, (UL) 1);

    for (p_i = tid; primes[p_i] < B; p_i += threads) {
      unsigned prime_ul = (UL) primes[p_i];

      mpz_set_lui(&p, prime_ul);

      mpz_set_lui(&tmp_2, (UL) (log((double) B) / log((double) prime_ul)));

      // tmp_2 = floor(log b / log p)
      mpz_powmod(&tmp, &p, &tmp_2, &n); // tmp = (p ** tmp_2) % n
      mpz_mult(&tmp_2, &tmp, &e);       // tmp_2 = tmp * e
      mpz_set(&e, &tmp_2);              // e = tmp_2
    }

    if (mpz_equal(&e, &MPZ_ONE)) continue;

    // try a variety of a values
    mpz_set_lui(&a, 2 + tid);

    for (it = 0; it < max_it; it ++) {
      //printf("it = %d\n", it);
      count ++;
      if (*finished) {
        //printf("Ran in %d iterations.\n", count);
        return;
      }

      // check for a freebie
      mpz_gcd(&d, &a, &n);
      if (mpz_lt(&MPZ_ONE, &d)) {
        *result = d;
        *finished = true;
        //printf("Ran in %d iterations.\n", count);
        return;
      }

      mpz_powmod(&b, &a, &e, &n);  // b = (a ** e) % n
      mpz_sub(&tmp, &b, &MPZ_ONE); // tmp = b - 1
      mpz_gcd(&d, &tmp, &n);       // d = gcd(tmp, n)

      // success!
      if (mpz_lt(&MPZ_ONE, &d) && mpz_lt(&d, &n)) {
        *result = d;
        *finished = true;
        //printf("Ran in %d iterations.\n", count);
        return;
      }

      // otherwise get a new value for a
#if 1
      mpz_mult(&tmp, &a, &a);               // tmp = a ** 2
      mpz_set_lui(&a, (UL) (i + it + tid)); // a = i + it + tid
      mpz_add(&tmp_2, &tmp, &a);            // tmp_2 = &tmp + a
      mpz_div(&tmp, &a, &tmp_2, &n);        // a = tmp_2 % n
#else
      mpz_add(&tmp, &a, &MPZ_ONE);
      mpz_set(&a, &tmp);
#endif
    }
  }
  // couldn't find anything... :(
  printf("Ran in %d iterations (and failed).\n", count);
}

int factorize(mpz_t *n, unsigned *primes, mpz_t *factor) {
  unsigned blocks = NUM_BLOCKS;
  unsigned threads_per_block = THREADS_PER_BLOCK;
  //unsigned threads = blocks * threads_per_block;

  size_t result_bytes = sizeof(mpz_t);

  mpz_t *d_result;
  bool *d_finished;
  if ((hipSuccess != hipMalloc((void **) &d_result, result_bytes)) ||
      (hipSuccess != hipMalloc((void **) &d_finished, sizeof(bool))) ||
      (hipSuccess != hipMemset(d_result, 0L, result_bytes)) ||
      (hipSuccess != hipMemset(d_finished, false, sizeof(bool)))) {
    fprintf(stderr, "Unable to allocate device memory!\n");
    return -1;
  }

  parallel_factorize_kernel<<<blocks, threads_per_block>>>
    (*n, primes, d_finished, d_result);

  if (hipSuccess != hipMemcpy(factor, d_result, result_bytes,
                                hipMemcpyDeviceToHost)) {
    fprintf(stderr, "Unable to retrieve result from host!\n");
    return -1;
  }

  hipFree(primes);
  hipFree(d_result);
  hipFree(d_finished);

  return 0;
}

void get_prime_table(unsigned *table, unsigned n) {
  primegen pg;

  primegen_init(&pg);

  for (unsigned i = 0; i < n; i ++) {
    table[i] = (unsigned) primegen_next(&pg);
  }
}

int generate_prime_table(unsigned **d_table) {
  /* approximately the number of unsigned 32-bit primes       */
  /* (actual number is 203,280,221)                           */
  /* paper claimed to use ~170,000,000 primes for experiments */
  /* may want to write this to disk at some point...          */
  unsigned primes = TABLE_SIZE;
  unsigned *h_table = (unsigned *) malloc(primes * sizeof(unsigned));
  if (NULL == h_table) {
    fprintf(stderr, "Unable to allocate host prime table!\n");
    return -1;
  }

  printf("Generating prime table... "); fflush(stdout);
  get_prime_table(h_table, primes);
  printf("done!\n"); fflush(stdout);

  /* move prime table to the gpu */
  printf("Transferring table to the gpu... "); fflush(stdout);
  if ((hipSuccess != hipMalloc((void **) d_table,
                                 primes * sizeof(unsigned))) ||
      (hipSuccess != hipMemcpy((void *) *d_table, (void *) h_table,
                                 primes * sizeof(unsigned),
                                 hipMemcpyHostToDevice))) {
    fprintf(stderr, "Unable to allocate device prime table!\n");
    return -1;
  }
  printf("done!\n"); fflush(stdout);

  free(h_table);

  return 0;
}
