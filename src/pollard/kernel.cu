#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"
#include "primegen.h"

#define NUM_BLOCKS 1
#define THREADS_PER_BLOCK 1

#define B_START 2
#define TABLE_SIZE (200 * 1000 * 1000)

__global__
void parallel_factorize_kernel(UL N, unsigned B, unsigned *primes,
                               bool *finished, mpz_t *results) {
  mpz_t n, a, d, p, e, b, tmp, tmp_2, MPZ_ONE;

  unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned threads = gridDim.x * blockDim.x;
  unsigned i = blockIdx.x * blockDim.x;

  mpz_init(&n);
  mpz_init(&a);
  mpz_init(&d);
  mpz_init(&p);
  mpz_init(&e);
  mpz_init(&b);
  mpz_init(&tmp);
  mpz_init(&tmp_2);

  mpz_init(&MPZ_ONE);
  mpz_set_i(&MPZ_ONE, 1);

  mpz_set_lui(&n, N);

  unsigned it;
  unsigned max_it = 80;

  mpz_set_lui(&e, (UL) 1);
  unsigned p_i;
  for (p_i = tid; primes[p_i] < B; p_i += threads) {
    mpz_set_lui(&p, (UL) primes[p_i]);
    // TODO: replace MPZ_ONE with logB / logp
    mpz_set_lui(&tmp_2, (UL) (log((double) B) / log((double) primes[p_i])));
                                      // tmp_2 = floor(log b / log p)
    mpz_powmod(&tmp, &p, &tmp_2, &n); // tmp = (p ** tmp_2) % n
    mpz_mult(&tmp_2, &tmp, &e);       // tmp_2 = tmp * e
    mpz_set(&e, &tmp_2);              // e = tmp_2
  }

  if (mpz_equal(&e, &MPZ_ONE)) return;

  // char *e_str = mpz_get_str(&e, NULL, 0);
  // printf("\tUsing e = %s\n", e_str);
  // free(e_str);

  // try a variety of a values
  mpz_set_lui(&a, 2 + tid);
  for (it = 0; it < max_it; it ++) {
    if (*finished) {
      return;
    }
    // char *a_str = mpz_get_str(&a, NULL, 0);
    // printf("\t\tUsing a = %s\n", a_str);
    // free(a_str);
    // check for a freebie
    mpz_gcd(&d, &a, &n);
    if (mpz_lt(&MPZ_ONE, &d)) {
      results[tid] = d;
      *finished = true;
      return;
    }

    mpz_powmod(&b, &a, &e, &n);  // b = (a ** e) % n
    mpz_sub(&tmp, &b, &MPZ_ONE); // tmp = b - 1
    mpz_gcd(&d, &tmp, &n);       // d = gcd(tmp, n)

    // success!
    if (mpz_lt(&MPZ_ONE, &d) && mpz_lt(&d, &n)) {
      results[tid] = d;
      *finished = true;
      return;
    }

    // otherwise get a new value for a
    mpz_mult(&tmp, &a, &a);               // tmp = a ** 2
    mpz_set_lui(&a, (UL) (i + it + tid)); // a = i + it + tid
    mpz_add(&tmp_2, &tmp, &a);            // tmp_2 = &tmp + a
    mpz_div(&tmp, &a, &tmp_2, &n);        // a = tmp_2 % n
  }
  // couldn't find anything... :(
}

int factorize(UL n, unsigned *primes, mpz_t *factor) {
  unsigned blocks = NUM_BLOCKS;
  unsigned threads_per_block = THREADS_PER_BLOCK;
  unsigned threads = blocks * threads_per_block;

  size_t results_bytes = threads * sizeof(mpz_t);

  mpz_t *d_results;
  if ((hipSuccess != hipMalloc((void **) &d_results, results_bytes)) ||
      (hipSuccess != hipMemset(d_results, 0L, results_bytes))) {
    fprintf(stderr, "Unable to allocate results table!\n");
    return -1;
  }

  // create global boolean used to exit on completion
  bool *d_finished;
  hipMalloc((void **) &d_finished, sizeof(bool));
  hipMemset(d_finished, false, sizeof(bool));

  unsigned B;
  unsigned max_B = ((n < TABLE_SIZE) ? n : TABLE_SIZE);
  for (B = B_START; B < max_B; B *= 2) {
    printf("Using B = %u\n", B);
    parallel_factorize_kernel<<<blocks, threads_per_block>>>
      (n, B, primes, d_finished, d_results);

    mpz_t *tmp_results = (mpz_t *) malloc(results_bytes);
    if (NULL == tmp_results) {
      fprintf(stderr, "Error allocating temporary result storage!\n");
      return -1;
    }
    if (hipSuccess != hipMemcpy(tmp_results, d_results, results_bytes,
                                  hipMemcpyDeviceToHost)) {
      fprintf(stderr, "Unable to retrieve results from host!\n");
      return -1;
    }

    mpz_t MPZ_ZERO;
    mpz_init(&MPZ_ZERO);
    mpz_set_i(&MPZ_ZERO, 0);

    unsigned thread;
    for (thread = 0; thread < threads; thread ++) {
      if (!mpz_equal(&MPZ_ZERO, &tmp_results[thread])) {
        mpz_set(factor, &tmp_results[thread]);
        return 0;
      }
    }
  }

  hipFree(primes);
  hipFree(d_results);
  hipFree(d_finished);

  return -1;
}

void get_prime_table(unsigned *table, unsigned n) {
  primegen pg;

  primegen_init(&pg);

  for (unsigned i = 0; i < n; i ++) {
    table[i] = (unsigned) primegen_next(&pg);
  }
}

int generate_prime_table(unsigned **d_table) {
  /* approximately the number of unsigned 32-bit primes       */
  /* (actual number is 203,280,221)                           */
  /* paper claimed to use ~170,000,000 primes for experiments */
  /* may want to write this to disk at some point...          */
  unsigned primes = TABLE_SIZE;
  unsigned *h_table = (unsigned *) malloc(primes * sizeof(unsigned));
  if (NULL == h_table) {
    fprintf(stderr, "Unable to allocate host prime table!\n");
    return -1;
  }

  printf("Generating prime table... "); fflush(stdout);
  get_prime_table(h_table, primes);
  printf("done!\n"); fflush(stdout);

  /* move prime table to the gpu */
  printf("Transferring table to the gpu... "); fflush(stdout);
  if ((hipSuccess != hipMalloc((void **) d_table,
                                 primes * sizeof(unsigned))) ||
      (hipSuccess != hipMemcpy((void *) *d_table, (void *) h_table,
                                 primes * sizeof(unsigned),
                                 hipMemcpyHostToDevice))) {
    fprintf(stderr, "Unable to allocate device prime table!\n");
    return -1;
  }
  printf("done!\n"); fflush(stdout);

  free(h_table);

  return 0;
}
